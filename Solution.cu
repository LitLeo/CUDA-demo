#include "hip/hip_runtime.h"
#include "Solution.h"

// 每个线程处理 stride 个数据
static __global__ void GA1Ker()
{
    // 计算线程ID
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;

 //    // 根据线程ID获得数据索引
 //    uint data_index = thread_index * stride;

	// // 越界判断
 //    if (data_index >= data_size)
 //        return; 
}

void Solution::training()
{
    hipEvent_t start, stop;
    float runTime = 0.0;
    float minRunTime = 1000.0f;
    dim3 _blocksize;
    _blocksize.x = this->blocksize.x; // = 32
    
    // block 大小从32-1024，以32为stride，得到最优block 大小
    for(_blocksize.y = 1; _blocksize.y <= 32; ++_blocksize.y) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // ga1.groupingAggregation1GPU();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&runTime, start, stop);
        if(runTime < minRunTime) {
            minRunTime = runTime;
            this->blocksize.y = _blocksize.y;
        }
    }
}

int Solution::solution()
{

    // 申请 device 端空间，并拷贝数据
    // checkCudaErrors(hipMalloc(&d_data, insize));
    // checkCudaErrors(hipMemcpy(d_data, h_data, insize, hipMemcpyHostToDevice));
    // checkCudaErrors(hipMalloc(&d_out_data, outsize));
    // checkCudaErrors(hipMemset(d_out_data, 0, outsize));

    

    // getLastCudaError("GA2_1Ker() execution failed");
    return 0;
}