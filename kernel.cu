#include "Solution.h"


int main(int argc, char const *argv[])
{
	if (argc == 1){
		cout << "usage error!  << endl";
	}

	// wampup函数，预热GPU，原理见 http://blog.csdn.net/litdaguang/article/details/50520549
	warmup();

    Solution so;
	hipEvent_t start, stop;
    float runTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// ga1.groupingAggregation1GPU();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&runTime, start, stop);

    return 0;
}



